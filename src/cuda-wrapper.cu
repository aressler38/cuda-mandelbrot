#include "cuda-wrapper.h"

/**
 * Host function that copies the data and launches the work on GPU
 */
unsigned *computeMandelbrot(
	complex_t *points,
	unsigned size,
	unsigned max_iterations=1024)
{
	static const int BLOCK_SIZE = 1024;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	unsigned *iterations = new unsigned[size];
	unsigned *device_iterations;
	complex_t *device_points;

	// Allocate memory on device
	hipMalloc((void **)&device_points, sizeof(complex_t)*size);
	hipMalloc((void **)&device_iterations, sizeof(unsigned)*size);
	// Copy the points to device
	hipMemcpy(device_points, points, sizeof(complex_t)*size, hipMemcpyHostToDevice);
	// Run kernel
	mandelbrot<<<blockCount, BLOCK_SIZE>>>(device_points, device_iterations, size, max_iterations);
	// Copy the iterations to host
	hipMemcpy(iterations, device_iterations, sizeof(unsigned)*size, hipMemcpyDeviceToHost);
	// Clean up 
	hipFree(device_iterations);
	hipFree(device_points);
	return iterations;
}
